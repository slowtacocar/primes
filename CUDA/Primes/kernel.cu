#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "math.h"

#include <iostream>
#include <chrono>

hipError_t addWithCuda(int* c, unsigned int size);

__global__ void primes(int* c)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    double sq = sqrt((double) n);
    if (n > 1) {
        for (int i = 2; i <= sq; i++) {
            if (n % i == 0) {
                return;
            }
        }
        atomicAdd(c, 1);
    }
}

int main()
{
    const int arraySize = 400;
    int c[1] = { 0 };

    // Add vectors in parallel.
    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
    hipError_t cudaStatus = addWithCuda(c, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    std::cout << *c << std::endl;
    std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start).count() << std::endl;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, unsigned int size)
{
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    primes << <size, 256 >> > (dev_c);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);

    return cudaStatus;
}
